#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define THREAD_PER_BLOCK 128

// Each block will compute the sum (reduce) of its section of the input array.
// The result from each block is written to the output array.
__global__ void reduce_v5(float *d_input, float *d_output)
{
    // Find the start of this block's data in the global array
    // subarray_size = 2 * blockDim.x
    float *input_begin = d_input + blockDim.x * blockIdx.x * 2;
    // Each thread sum two elements from global memory into a register 
    float sum = input_begin[threadIdx.x] + input_begin[threadIdx.x + blockDim.x];
    
    // Shuffle within each warp to reduce the sum
    sum += __shfl_down_sync(0xffffffff, sum, 16);
    sum += __shfl_down_sync(0xffffffff, sum, 8);
    sum += __shfl_down_sync(0xffffffff, sum, 4);
    sum += __shfl_down_sync(0xffffffff, sum, 2);
    sum += __shfl_down_sync(0xffffffff, sum, 1);

    // initialize shared memory
    __shared__ float warpLevelSums[32];
    // calculate landId and WarpId
    const int warpId = threadIdx.x /32;
    const int laneId = threadIdx.x % 32;
    // store the reduced sum for each warp in the shared memory
    if (laneId == 0)
        warpLevelSums[warpId] = (warpId < blockDim.x / 32) ? sum : 0.f;
    
    __syncthreads();

    // Shuffle the first warp
    if (warpId == 0)
    {
        sum = warpLevelSums[laneId];
        sum += __shfl_down_sync(0xffffffff, sum, 16);
        sum += __shfl_down_sync(0xffffffff, sum, 8);
        sum += __shfl_down_sync(0xffffffff, sum, 4);
        sum += __shfl_down_sync(0xffffffff, sum, 2);
        sum += __shfl_down_sync(0xffffffff, sum, 1);
    }

    
    // Only the first thread in each block writes the final sum to the output array
    if (threadIdx.x == 0)
        d_output[blockIdx.x] = sum;
}

// Compare two arrays for near-equality; returns true if all elements are nearly the same
bool check(float *out, float *res, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (abs(out[i] - res[i]) > 0.005)
            return false;
    }
    return true;
}

int main()
{
    // Number of input elements to reduce
    const int N = 32 * 1024 * 1024;

    // --- Memory allocation on CPU ---
    float *input = (float *)malloc(N * sizeof(float));

    // --- Memory allocation on GPU ---
    float *d_input;
    hipMalloc((void **)&d_input, N * sizeof(float));

    // Number of blocks: each block reduces THREAD_PER_BLOCK elements
    int block_num = N / (2 * THREAD_PER_BLOCK);

    // Size of each subarray that each block will process
    int subarray_size = N / block_num;

    // Allocate memory for block-wise reduction results on CPU
    float *output = (float *)malloc(block_num * sizeof(float));

    // Allocate memory for block-wise reduction results on GPU
    float *d_output;
    hipMalloc((void **)&d_output, block_num * sizeof(float));

    // Allocate memory for CPU-computed reference result
    float *result = (float *)malloc(block_num * sizeof(float));

    // Initialize input array with random values between -1 and 1
    for (int i = 0; i < N; i++)
    {
        input[i] = 2.0 * (float)drand48() - 1.0;
    }

    // --- CPU computation of per-block reductions (reference answer) ---
    for (int i = 0; i < block_num; i++)
    {
        float cur = 0;
        for (int j = 0; j < subarray_size; j++)
        {
            cur += input[i * subarray_size + j];
        }
        result[i] = cur;
    }

    // Copy input data from CPU to GPU
    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    // Configure CUDA kernel launch: one block per chunk, THREAD_PER_BLOCK threads per block
    dim3 Grid(block_num, 1);
    dim3 Block(THREAD_PER_BLOCK, 1);

    // Launch reduction kernel
    reduce_v5<<<Grid, Block>>>(d_input, d_output);

    // Copy the per-block sums from GPU back to CPU
    hipMemcpy(output, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost);

    // Compare GPU and CPU results
    if (check(output, result, block_num))
        printf("The answer is correct\n");
    else
    {
        printf("The answer is wrong\n");
        for (int i = 0; i < block_num; i++)
        {
            printf("%lf ", output[i]);
        }
        printf("\n");
    }

    // Free GPU memory
    hipFree(d_input);
    hipFree(d_output);

    // Free CPU memory
    free(input);
    free(output);
    free(result);

    return 0;
}