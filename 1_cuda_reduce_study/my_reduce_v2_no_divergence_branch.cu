#include <cstdio>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define THREAD_PER_BLOCK 256

// Each block will compute the sum (reduce) of its section of the input array.
// The result from each block is written to the output array.
__global__ void reduce_v2(float *d_input, float *d_output)
{
    // Find the start of this block's data in the global array
    float *input_begin = d_input + blockDim.x * blockIdx.x;
    // Initialize shared memory for this block
    __shared__ float input_shared[THREAD_PER_BLOCK];
    input_shared[threadIdx.x] = input_begin[threadIdx.x];
    __syncthreads(); // Ensure all threads have written their data to shared memory

    // if (threadIdx.x == 0, 1, 2, 3)
    //    input_shared[threadIdx.x * 2] += input_shared[threadIdx.x * 2 + 1];
    // if (threadIdx.x == 0, 1)
    //     input_shared[threadIdx.x * 4] += input_shared[threadIdx.x * 4 + 2];
    // if (threadIdx.x == 0)
    //     input_shared[threadIdx.x * 8] += input_shared[threadIdx.x * 8 + 4];
    // Parallel reduction: repeatedly halve the number of participating threads
    // On each step, threads with indices less than blockDim.x/(2*i) add the value 
    // at index threadIdx.x*2*i to the value at threadIdx.x*2*i + i
    for (int i = 1; i < blockDim.x; i *= 2)
    {
        if (threadIdx.x <  blockDim.x / (i * 2))
        {   
            int index = threadIdx.x * 2 * i;
            input_shared[index] += input_shared[index + i];
        }
        __syncthreads(); // Synchronize to make sure all threads are done before the next step
    }
    // Only the first thread in each block writes the final sum to the output array
    if (threadIdx.x == 0)
        d_output[blockIdx.x] = input_shared[0];
}

// Compare two arrays for near-equality; returns true if all elements are nearly the same
bool check(float *out, float *res, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (abs(out[i] - res[i]) > 0.005)
            return false;
    }
    return true;
}

int main()
{
    // Number of input elements to reduce
    const int N = 32 * 1024 * 1024;

    // --- Memory allocation on CPU ---
    // Q: Why do we use a pointer to allocate memory? Is input here a pointer?
    float *input = (float *)malloc(N * sizeof(float));

    // --- Memory allocation on GPU ---
    float *d_input;
    hipMalloc((void **)&d_input, N * sizeof(float));

    // Number of blocks: each block reduces THREAD_PER_BLOCK elements
    int block_num = N / THREAD_PER_BLOCK;

    // Allocate memory for block-wise reduction results on CPU
    float *output = (float *)malloc(block_num * sizeof(float));

    // Allocate memory for block-wise reduction results on GPU
    float *d_output;
    hipMalloc((void **)&d_output, block_num * sizeof(float));

    // Allocate memory for CPU-computed reference result
    float *result = (float *)malloc(block_num * sizeof(float));

    // Initialize input array with random values between -1 and 1
    for (int i = 0; i < N; i++)
    {
        input[i] = 2.0 * (float)drand48() - 1.0;
    }

    // --- CPU computation of per-block reductions (reference answer) ---
    for (int i = 0; i < block_num; i++)
    {
        float cur = 0;
        for (int j = 0; j < THREAD_PER_BLOCK; j++)
        {
            cur += input[i * THREAD_PER_BLOCK + j];
        }
        result[i] = cur;
    }

    // Copy input data from CPU to GPU
    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    // Configure CUDA kernel launch: one block per chunk, THREAD_PER_BLOCK threads per block
    dim3 Grid(N / THREAD_PER_BLOCK, 1);
    dim3 Block(THREAD_PER_BLOCK, 1);

    // Launch reduction kernel
    reduce_v2<<<Grid, Block>>>(d_input, d_output);

    // Copy the per-block sums from GPU back to CPU
    hipMemcpy(output, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost);

    // Compare GPU and CPU results
    if (check(output, result, block_num))
        printf("The answer is correct\n");
    else
    {
        printf("The answer is wrong\n");
        for (int i = 0; i < block_num; i++)
        {
            printf("%lf ", output[i]);
        }
        printf("\n");
    }

    // Free GPU memory
    hipFree(d_input);
    hipFree(d_output);

    // Free CPU memory
    free(input);
    free(output);
    free(result);

    return 0;
}