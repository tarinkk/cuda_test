#include <cstdio>               // printf
#include <hip/hip_runtime.h>
#include <stdlib.h>             // malloc, free, drand48, abs
       // cudaMalloc, cudaMemcpy

// Utility: Fill a host matrix with random numbers in the range [-1, 1]
void random_matrix(int M, int N, float *A_ptr)
{
    for (int m = 0; m < M; m++)
        for(int n = 0; n < N; n++)
        {
            A_ptr[m * N + n] = 2.0 * (float)drand48() - 1.0;
        } 
}

// Reference SGEMM on CPU: triple-nested loop (O(M.N.K))
void cpu_sgemm(float *A_ptr, float *B_ptr, float *C_ptr, int M, int N, int K)
{
    for (int m = 0; m < M; m++)
        for(int n = 0; n < N; n++)
        {
            for(int k = 0; k < K; k++)
            {
                C_ptr[m * N + n] += A_ptr[m * K + k] * B_ptr[k * N + n];
            }
        }
}

// Compare two matrices element-wise; return max difference
float compare_matrices(int M, int N, float *C_gpu, float *C_cpu)
{
    float diff = 0.0f;
    float max_diff = 0.0f;
    for (int m = 0; m < M; m++)
        for(int n = 0; n < N; n++)
        {
            diff = abs(C_gpu[m * N + n] - C_cpu[m * N + n]);
            if (diff > max_diff)
            {
                max_diff = diff;
            }
        }
    return max_diff;
}

#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])

template <unsigned int C_TILE_SIZE, 
          unsigned int K_TILE_SIZE, 
          unsigned int NUM_PER_THREAD>
__global__ void cuda_sgemm_v3(float *A_ptr, float *B_ptr, float *C_ptr, const int M, const int N, const int K)
{
    // Thread & block coordinates
    const int tx = threadIdx.x;                 // local column id inside the block 
    const int ty = threadIdx.y;                 // local row id  inside the block
    const int xBase = blockIdx.x * C_TILE_SIZE; // global col of block’s left edge
    const int yBase = blockIdx.y * C_TILE_SIZE; // global row of block’s top edge

    // shared memory staging buffers
    __shared__ float a_shared[C_TILE_SIZE][K_TILE_SIZE];
    __shared__ float b_shared[K_TILE_SIZE][C_TILE_SIZE];

    // per-thread accumulator
    float temp[NUM_PER_THREAD] = {0.0};

    // Total number of K tiles (ceil division)
    const int NUM_TILES = (K + K_TILE_SIZE - 1) / K_TILE_SIZE;

    // Loop over K tiles
    for (int tile = 0; tile < NUM_TILES; ++tile)
    {  
        const int kBase = tile * K_TILE_SIZE; // starting K index of this tile
        
        // Load A tile
        for (int kk = 4 * tx; kk < K_TILE_SIZE; kk += NUM_PER_THREAD * blockDim.x)
        {
            int k = kBase + kk;     //global k-index in A
            int y = yBase + ty;     //global y-index in A
            if (k < K && y < M)
                FETCH_FLOAT4(a_shared[ty][kk]) = FETCH_FLOAT4(A_ptr[y * K + k]);
        }  

        // Load B tile
        for (int kk = ty; kk < K_TILE_SIZE; kk += blockDim.y) 
        {
            int xx = 4 * tx;        // Local col in B tile
            int k = kBase + kk;     // global k-index in B
            int x = xBase + xx;     // global x-index in B
            if (k < K && x < N)
                FETCH_FLOAT4(b_shared[kk][xx]) = FETCH_FLOAT4(B_ptr[k * N + x]); 
        }

        __syncthreads();   // ensures tiles fully populated

        // Compute the partial dot product for this tile
        for (int kk = 0; kk < K_TILE_SIZE; ++kk)
            for (int i = 0; i < NUM_PER_THREAD; ++i)
                {
                    int yy = ty;
                    int xx = 4 * tx + i;
                    int x = xBase + xx;
                    int y = yBase + yy;
                    int k = kBase + kk;
                    if (k < K && y < M && x < N)
                        temp[i] += a_shared[yy][kk] * b_shared[kk][xx]; 
                }
        __syncthreads(); // avoid data hazard before next load
    }

    // Write back the result in C

    for (int i = 0; i < NUM_PER_THREAD; ++i)
    {
        int y = yBase + ty;
        int x = xBase + 4 * tx + i;
        if (y < M && x < N) C_ptr[y * N + x] = temp[i];
    }
    

}

int main()
{
    // Matrix sizes
    int M = 512;
    int N = 512;
    int K = 512;

    // Host memory allocation
    const size_t mem_size_A = M * K * sizeof(float);
    const size_t mem_size_B = K * N * sizeof(float);
    const size_t mem_size_C = M * N * sizeof(float);
    float *matrix_A_host = (float *)malloc(mem_size_A);
    float *matrix_B_host = (float *)malloc(mem_size_B);
    float *matrix_C_host_gpu_calc = (float *)malloc(mem_size_C);
    float *matrix_C_host_cpu_calc = (float *)malloc(mem_size_C);

    // Generate random inputs
    random_matrix(M, K, matrix_A_host);
    random_matrix(K, N, matrix_B_host);
    printf("Random matrices generated.\n");

    // Initialize output matrices to zero
    memset(matrix_C_host_gpu_calc, 0, mem_size_C);
    memset(matrix_C_host_cpu_calc, 0, mem_size_C);
    printf("Memory allocated and initialized.\n");

    // Device memory allocation
    float *matrix_A_device, *matrix_B_device, *matrix_C_device;
    hipMalloc((void **)&matrix_A_device, mem_size_A);
    hipMalloc((void **)&matrix_B_device, mem_size_B);
    hipMalloc((void **)&matrix_C_device, mem_size_C);

    // Copy inputs to GPU
    hipMemcpy(matrix_A_device, matrix_A_host, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(matrix_B_device, matrix_B_host, mem_size_B, hipMemcpyHostToDevice);
    printf("Data copied to device.\n");

    // CPU (reference) computation
    cpu_sgemm(matrix_A_host, matrix_B_host, matrix_C_host_cpu_calc, M, N, K);
    printf("CPU SGEMM completed.\n");
    
    // Launch GPU SGEMM kernel
    constexpr int N_BLOCK_SIZE = 8;
    constexpr int M_BLOCK_SIZE = 32;
    // for simplicity, we assume K_TILE_SIZE is a multiple of 4
    constexpr int K_TILE_SIZE = 64;
    constexpr int C_TILE_SIZE = 32;
    constexpr int NUM_PER_THREAD = 4;;
    dim3 block(N_BLOCK_SIZE, M_BLOCK_SIZE);
    dim3 grid((N + C_TILE_SIZE - 1) / C_TILE_SIZE, (M + C_TILE_SIZE - 1) / C_TILE_SIZE);
    cuda_sgemm_v3<C_TILE_SIZE, K_TILE_SIZE, NUM_PER_THREAD><<<grid, block>>>(matrix_A_device, matrix_B_device, matrix_C_device, M, N, K);
    printf("GPU SGEMM kernel launched.\n");

    // Retrieve result from GPU
    hipMemcpy(matrix_C_host_gpu_calc, matrix_C_device, mem_size_C, hipMemcpyDeviceToHost);
    printf("Data copied from device to host.\n");

    // Verify results
    float diff = compare_matrices(M, N, matrix_C_host_gpu_calc, matrix_C_host_cpu_calc);
    printf("Comparison of GPU and CPU results completed.\n");

    if (diff > 0.5f)
    {
        printf("Error: GPU and CPU results do not match! Difference: %f\n", diff);

    }
    else
    {
        printf("Success: GPU and CPU results match! Difference: %f\n", diff);
    }

    // Cleanup
    free(matrix_A_host);
    free(matrix_B_host);
    free(matrix_C_host_cpu_calc);
    free(matrix_C_host_gpu_calc);

    hipFree(matrix_A_device);
    hipFree(matrix_B_device);
    hipFree(matrix_C_device);
    printf("Memory freed.\n");
    return 0;
}