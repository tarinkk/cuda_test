#include <cstdio>               // printf
#include <hip/hip_runtime.h>
#include <stdlib.h>             // malloc, free, drand48, abs
       // cudaMalloc, cudaMemcpy

// Utility: Fill a host matrix with random numbers in the range [-1, 1]
void random_matrix(int M, int N, float *A_ptr)
{
    for (int m = 0; m < M; m++)
        for(int n = 0; n < N; n++)
        {
            A_ptr[m * N + n] = 2.0 * (float)drand48() - 1.0;
        } 
}

// Reference SGEMM on CPU: triple-nested loop (O(M.N.K))
void cpu_sgemm(float *A_ptr, float *B_ptr, float *C_ptr, int M, int N, int K)
{
    for (int m = 0; m < M; m++)
        for(int n = 0; n < N; n++)
        {
            for(int k = 0; k < K; k++)
            {
                C_ptr[m * N + n] += A_ptr[m * K + k] * B_ptr[k * N + n];
            }
        }
}

// Compare two matrices element-wise; return max difference
float compare_matrices(int M, int N, float *C_gpu, float *C_cpu)
{
    float diff = 0.0f;
    float max_diff = 0.0f;
    for (int m = 0; m < M; m++)
        for(int n = 0; n < N; n++)
        {
            diff = abs(C_gpu[m * N + n] - C_cpu[m * N + n]);
            if (diff > max_diff)
            {
                max_diff = diff;
            }
        }
    return max_diff;
}

template <unsigned int C_TILE_SIZE, unsigned int K_TILE_SIZE, unsigned int STRIDE>
__global__ void cuda_sgemm_v2(float *A_ptr, float *B_ptr, float *C_ptr, const int M, const int N, const int K)
{
    const int tx = threadIdx.x;                 // local column id inside the block 
    const int ty = threadIdx.y;                 // local row id  inside the block
    const int xBase = blockIdx.x * C_TILE_SIZE; // global col of block’s left edge
    const int yBase = blockIdx.y * C_TILE_SIZE; // global row of block’s top edge

    // Pointers to the start of the A and B matrices for this block.
    float *A_ptr_start = A_ptr + yBase * K;
    float *B_ptr_start = B_ptr + xBase;

    // shared memory tiles
    __shared__ float a_shared[C_TILE_SIZE][K_TILE_SIZE];
    __shared__ float b_shared[K_TILE_SIZE][C_TILE_SIZE];

    // running sum for C[y][x]
    float temp[STRIDE][STRIDE] = {0.0};

    // Total number of K tiles (ceil division)
    const int NUM_TILES = (K + K_TILE_SIZE - 1) / K_TILE_SIZE;

    // Loop over K tiles
    for (int tile = 0; tile < NUM_TILES; ++tile)
    {  
        const int kBase = tile * K_TILE_SIZE; // starting K index of this tile
        
        // Load A tile
        for (int kk = tx; kk < K_TILE_SIZE; kk += blockDim.x)
            for (int yy = ty; yy < C_TILE_SIZE; yy += blockDim.y)
            {
                int k = kBase + kk;
                int y = yBase + yy;
                a_shared[yy][kk] = (y < M && k < K) ? A_ptr_start[yy * K + k] : 0.0f;
            }

        // Load B tile
        for (int kk = threadIdx.y; kk < K_TILE_SIZE; kk += blockDim.y) 
            for (int xx = tx; xx < C_TILE_SIZE; xx += blockDim.x)
            {
                int k = kBase + kk;
                int x = xBase + xx;
                b_shared[kk][xx] = (k < K && x < N) ? B_ptr_start[k * N + xx] : 0.0f; 
            }

        __syncthreads();   // ensures tiles fully populated

        // Compute the partial dot product for this tile
        for (int kk = 0; kk < K_TILE_SIZE; ++kk)
            for (int ry = 0; ry < STRIDE; ++ry)
                for (int cx = 0; cx < STRIDE; ++cx)
                {
                    int yy = ty + ry * blockDim.y;
                    int xx = tx + cx * blockDim.x;
                    temp[ry][cx] += a_shared[yy][kk] * b_shared[kk][xx]; 
                }
        __syncthreads(); // avoid data hazard before next load
    }

    // Write back the result in C

    for (int ry = 0; ry < STRIDE; ++ry)
        for (int cx = 0; cx < STRIDE; ++cx)
        {
            int y = yBase + ty + ry * blockDim.y;
            int x = xBase + tx + cx * blockDim.x;
            if (y < M && x < N) C_ptr[y * N + x] = temp[ry][cx];
        }
    

}

int main()
{
    // Matrix sizes
    int M = 512;
    int N = 512;
    int K = 512;

    // Host memory allocation
    const size_t mem_size_A = M * K * sizeof(float);
    const size_t mem_size_B = K * N * sizeof(float);
    const size_t mem_size_C = M * N * sizeof(float);
    float *matrix_A_host = (float *)malloc(mem_size_A);
    float *matrix_B_host = (float *)malloc(mem_size_B);
    float *matrix_C_host_gpu_calc = (float *)malloc(mem_size_C);
    float *matrix_C_host_cpu_calc = (float *)malloc(mem_size_C);

    // Generate random inputs
    random_matrix(M, K, matrix_A_host);
    random_matrix(K, N, matrix_B_host);
    printf("Random matrices generated.\n");

    // Initialize output matrices to zero
    memset(matrix_C_host_gpu_calc, 0, mem_size_C);
    memset(matrix_C_host_cpu_calc, 0, mem_size_C);
    printf("Memory allocated and initialized.\n");

    // Device memory allocation
    float *matrix_A_device, *matrix_B_device, *matrix_C_device;
    hipMalloc((void **)&matrix_A_device, mem_size_A);
    hipMalloc((void **)&matrix_B_device, mem_size_B);
    hipMalloc((void **)&matrix_C_device, mem_size_C);

    // Copy inputs to GPU
    hipMemcpy(matrix_A_device, matrix_A_host, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(matrix_B_device, matrix_B_host, mem_size_B, hipMemcpyHostToDevice);
    printf("Data copied to device.\n");

    // CPU (reference) computation
    cpu_sgemm(matrix_A_host, matrix_B_host, matrix_C_host_cpu_calc, M, N, K);
    printf("CPU SGEMM completed.\n");
    
    // Launch GPU SGEMM kernel
    constexpr int BLOCK_SIZE = 16;
    // for simplicity, we assume K are multiple of K_TILE_SIZE and K_TILE_SIZE is a multiple of BLOCK_SIZE
    constexpr int K_TILE_SIZE = 64;
    constexpr int C_TILE_SIZE = 32;
    constexpr int STRIDE = C_TILE_SIZE / BLOCK_SIZE;
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((N + C_TILE_SIZE - 1) / C_TILE_SIZE, (M + C_TILE_SIZE - 1) / C_TILE_SIZE );
    cuda_sgemm_v2<C_TILE_SIZE, K_TILE_SIZE, STRIDE><<<grid, block>>>(matrix_A_device, matrix_B_device, matrix_C_device, M, N, K);
    printf("GPU SGEMM kernel launched.\n");

    // Retrieve result from GPU
    hipMemcpy(matrix_C_host_gpu_calc, matrix_C_device, mem_size_C, hipMemcpyDeviceToHost);
    printf("Data copied from device to host.\n");

    // Verify results
    float diff = compare_matrices(M, N, matrix_C_host_gpu_calc, matrix_C_host_cpu_calc);
    printf("Comparison of GPU and CPU results completed.\n");

    if (diff > 0.5f)
    {
        printf("Error: GPU and CPU results do not match! Difference: %f\n", diff);

    }
    else
    {
        printf("Success: GPU and CPU results match! Difference: %f\n", diff);
    }

    // Cleanup
    free(matrix_A_host);
    free(matrix_B_host);
    free(matrix_C_host_cpu_calc);
    free(matrix_C_host_gpu_calc);

    hipFree(matrix_A_device);
    hipFree(matrix_B_device);
    hipFree(matrix_C_device);
    printf("Memory freed.\n");
    return 0;
}